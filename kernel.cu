﻿#include "NestedMonteCarloVaR.cuh"

int main()
{
	const int exp_times = 10;   // Total times of MC

	const int path_ext = 1024;  // Number of the outer MC loops
	const int path_int = 1024;  // Number of the inner MC loops

	cout << endl << "== SET UP ==" << endl;
	cout << "Experiment Times: " << exp_times << endl;
	cout << "Path External: " << path_ext << endl;
	cout << "Path Internal: " << path_int << endl;

	cout << endl << "== DEVICE ==" << endl;
	//cout << "CPU" << endl;

	int deviceCount;

	CUDA_CALL(hipGetDeviceCount(&deviceCount), "GetDeviceCount", __FILE__, __LINE__);

	printf("Number of CUDA devices %d.\n", deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;

		CUDA_CALL(hipGetDeviceProperties(&deviceProp, dev), "Get Device Properties", __FILE__, __LINE__);

		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				cout << "No CUDA GPU has been detected\n";
				return -1;
			}
			else if (deviceCount == 1) {
				cout << "There is 1 device supporting CUDA\n";
			}
			else {
				cout << "There are " << deviceCount << " devices supporting CUDA\n";
			}
		}

		printf("For device #%d\n", dev);
		printf("Device name:                %s\n", deviceProp.name);
		printf("Major revision number:      %d\n", deviceProp.major);
		printf("Minor revision Number:      %d\n", deviceProp.minor);
		printf("Total Global Memory:        %d\n", deviceProp.totalGlobalMem);
		printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerBlock);
		printf("Total const mem size:       %d\n", deviceProp.totalConstMem);
		printf("Warp size:                  %d\n", deviceProp.warpSize);
		printf("Maximum block dimensions:   %d x %d x %d\n", deviceProp.maxThreadsDim[0], \
			deviceProp.maxThreadsDim[1], \
			deviceProp.maxThreadsDim[2]);

		printf("Maximum grid dimensions:    %d x %d x %d\n", deviceProp.maxGridSize[0], \
			deviceProp.maxGridSize[1], \
			deviceProp.maxGridSize[2]);
		printf("Clock Rate:                 %d\n", deviceProp.clockRate);
		printf("Number of muliprocessors:   %d\n", deviceProp.multiProcessorCount);

	}


	const int var_t = 1;					// VaR duration
	const float var_per = 0.95f;				// 1-percentile

	const int port_n = 4;					// Number of products in the portfolio
	//float port_w[port_n] = { 0.0f, 0.0f, 0.0f, 1.0f };		// Weights of the products in the portfolio
	float port_w[port_n] = { 0.3f, 0.3f, 0.1f, 0.3f };		// Weights of the products in the portfolio
														// { bond, stock, basket option, barrier option}
	const float risk_free = 0.02f;

	const float bond_par = 1000.0f;			// Par value of bond
	const float bond_c = 100.0f;			// Coupon
	const int bond_m = 10;					// Maturity
	float bond_y[bond_m] = {
			5.00f, 5.69f, 6.09f, 6.38f, 6.61f,
			6.79f, 6.94f, 7.07f, 7.19f, 7.30f
	};										// yeild curve
	const float sigma = 1.5f;				// sigma

	const float stock_s0 = 300.0f;			// Start value of stock
	const float stock_mu = risk_free;			// risk free(or mean)
	const float stock_var = 0.13f;			// Volatility
	const int stock_x = 100;					// Number of shares

	Stock* s1 = new Stock(stock_s0, stock_mu, stock_var, 100);
	Stock* s2 = new Stock(stock_s0, stock_mu, stock_var, 100);
	const int bskop_n = 2;								// Number of stocks in basket
	const float bskop_k = 3100.0f;						// Execution price
	const int bskop_t = 1;								// Maturity of basket option
	Stock bskop_stocks[bskop_n] = { *s1, *s2 };			// List of stocks
	float bskop_cov[bskop_n * bskop_n] = { 1.0f, 0.5f,
										   0.5f, 1.0f };	// Covariance matrix
	float bskop_w[bskop_n] = { 0.5f, 0.5f };				// weight

	const float barop_k = 310.0f;				// Execution price
	const float barop_h = 320.0f;				// Barrier
	const int barop_t = 30;						// Maturity(steps of inner path)

	NestedMonteCarloVaR* mc = new NestedMonteCarloVaR(
		path_ext, path_int,
		var_t, var_per,
		port_n, port_w,
		risk_free
	);
	mc->bond_init(bond_par, bond_c, bond_m, bond_y, sigma, 0);
	mc->stock_init(stock_s0, stock_mu, stock_var, stock_x, 1);
	mc->bskop_init(bskop_n, bskop_stocks, bskop_cov, bskop_k, bskop_w, bskop_t, 2);
	mc->barop_int(s1, barop_k, barop_h, barop_t, 3);

	cout << endl << "== EXECUTION ==" << endl;

	// Warm up
	mc->execute();

	double exe_time = 0.0;
	for (int i = 0; i < exp_times; i++) {
		exe_time += mc->execute();
		cout << "Experiment # " << i << " finished." << endl;
	}

	cout << endl << "== RESULT ==" << endl;
	cout << "AVG EXECUTION TIME: " << exe_time / exp_times << " ms." << endl;
	return 0;
}